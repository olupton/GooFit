#include "hip/hip_runtime.h"
#include "ResonancePdf.hh" 

__device__ fptype twoBodyCMmomSq(fptype rMassSq, fptype d1m, fptype d2m)
{
  // Define something to return the momentum squared so we can defer
  // deciding what to do with unphysical momenta...
  return 0.25 * rMassSq * (1.0 - ((d1m+d2m)*(d1m+d2m) / rMassSq)) * (1.0 - ((d1m - d2m)*(d1m - d2m)/rMassSq));
}

__device__ fptype twoBodyCMmom (fptype rMassSq, fptype d1m, fptype d2m) {
  // For A -> B + C, calculate momentum of B and C in rest frame of A. 
  // PDG 38.16.

  //fptype kin1 = 1 - POW(d1m+d2m, 2) / rMassSq;
  //if (kin1 > 0)
  //  kin1 = SQRT(kin1);
  //else
  //  kin1 = 1;
  
  //fptype kin2 = 1 - POW(d1m-d2m, 2) / rMassSq;
  //if (kin2 > 0)
  //  kin2 = SQRT(kin2);
  //else
  //  kin2 = 1;

  //return 0.5*SQRT(rMassSq)*kin1*kin2; 
  return SQRT(twoBodyCMmomSq(rMassSq, d1m, d2m));
}

__device__ fptype bachelorMomSq(fptype otherMass, fptype motherMass, fptype bachelorMass)
{
  fptype
    motherMassSq(motherMass*motherMass),
    massSumSq((otherMass + bachelorMass)*(otherMass + bachelorMass)),
    massDiffSq((otherMass- bachelorMass)*(otherMass - bachelorMass));
  return 0.25 * (motherMassSq - massSumSq) * (motherMassSq - massDiffSq) / (otherMass * otherMass);
}

// For D -> (R -> AB)C calculate momentum of C (and D) in rest frame of R (A+B)
// otherMass is m_R, motherMass is m_D and bachelorMass is m_C
__device__ fptype bachelorMom(fptype otherMass, fptype motherMass, fptype bachelorMass)
{
  // This will sometimes get called with masses which are kinematically forbidden (i.e. m_R + m_C > m_D)
  // In these cases we just want to return something of the right order of magnitude.
  // The way in which we do this is just copied from the twoBodyCMmom() function above.
  //fptype
  //kin1(1.0 - POW((otherMass + bachelorMass)/motherMass, 2.0)),
  //  kin2(1.0 - POW((otherMass - bachelorMass)/motherMass, 2.0));
  //if(kin1 >= 0.0)
  //  kin1 = SQRT(kin1);
  //else
  //  kin1 = SQRT(-kin1);//kin1 = 1.0;
  //kin1 = SQRT(FMAX(0.0, kin1));//(FABS(kin1));
  
  // BIG FAT WARNING
  // try and emulate what MINT does (did) here, by using the absolute value of 'kin2' if it is negative
  // all choices here are quite arbitrary....
  //if(kin2 >= 0.0)
  //  kin2 = SQRT(kin2);
  //else
  //  kin2 = SQRT(-kin2);//1.0;
  //kin2 = SQRT(FMAX(0.0, kin2));
  
  //return FMAX(1e-6, 0.5 * POW(motherMass, 2.0) * kin1 * kin2 / otherMass);
  return SQRT(bachelorMomSq(otherMass, motherMass, bachelorMass));
}

__device__ fptype dampingFactorSquare (fptype cmmom, int spin, fptype mRadius)
{
  fptype square = mRadius*mRadius*cmmom*cmmom;
  fptype dfsq = 1 + square; // This accounts for spin 1
  if (2 == spin)
    dfsq += 8 + 2*square + square*square; // Coefficients are 9, 3, 1.

  // Spin 3 and up not accounted for. 
  return dfsq; 
}

__device__ fptype unNormalisedDampingFactorSquare(fptype measured_momentum, int spin, fptype meson_radius)
{
  printf("WARNING: you are using an incomplete, untested damping factor implementation!\n");
  if(spin == 0)
    return 1.0; 
  fptype
    square(measured_momentum * measured_momentum * meson_radius * meson_radius),
    ret(0.0);
  if(spin == 1)
    ret = 2.0 * square / (1.0 + square);
  if(spin == 2)
    ret = 13.0 * square * square / ( ((square - 3.0)*(square - 3.0)) + 9.0*square);
  return ret;
}

__device__ fptype dampingFactorRatioSquare(fptype nummom, fptype denmom, int spin, fptype meson_radius, bool useUnNormalisedFactor)
{
  if(useUnNormalisedFactor)
    return unNormalisedDampingFactorSquare(denmom, spin, meson_radius);
  else
    return dampingFactorSquare(nummom, spin, meson_radius) / dampingFactorSquare(denmom, spin, meson_radius);
}

// For spin 1:
// (p_D + p_C)_mu( -g_munu + P_mu P_nu * massFactor)(p_B - p_A)_nu where P = p_A + p_B
// = -(p_A + p_C + p_B + p_C).(p_B + p_C - p_A - p_C) + massFactor*(p_D + p_C).(p_D - p_C)*(p_A + p_B).(p_B - p_A)
// = -(s_BC - s_AC) + massFactor*(m_D^2 - m_C^2)*(m_B^2 - m_A^2)
// = -{ (s_BC - s_AC) + massFactor*(m_D^2 - m_C^2)*(m_A^2 - m_B^2) }
//
// For spin 2:
// (p_D + p_C)_mu(p_D + p_C)_nu T_mu,nu,alpha,beta (p_B - p_A)_alpha(p_B - p_A)_beta
// where T_mu,nu,alpha,beta = (1/2)*(T_mu,alpha*T_nu,beta + T_mu,beta*T_nu,alpha) - (1/3)T_mu,nu*T_alpha,beta
// and T_mu,nu = -g_mu,nu + massFactor*P_mu,P_nu
__device__ fptype spinFactorABC(unsigned int spin, fptype motherMass, fptype _mA, fptype _mB, fptype _mC, fptype _mAB, fptype _mAC, fptype _mBC, fptype massFactor)
{
  if(spin == 0)
    return 1.0;
  fptype sFactor(-1.0);
  sFactor *= ((_mBC - _mAC) + (massFactor*(motherMass*motherMass - _mC*_mC)*(_mA*_mA-_mB*_mB)));
  if (2 == spin) {
    sFactor *= sFactor;
    fptype extraterm = ((_mAB-(2*motherMass*motherMass)-(2*_mC*_mC))+massFactor*pow((motherMass*motherMass-_mC*_mC),2));
    extraterm *= ((_mAB-(2*_mA*_mA)-(2*_mB*_mB))+massFactor*pow((_mA*_mA-_mB*_mB),2));
    extraterm /= 3;
    sFactor -= extraterm;
  }
  return sFactor;
}

__device__ fptype spinFactor (unsigned int spin, fptype motherMass, fptype daug1Mass, fptype daug2Mass, fptype daug3Mass, fptype m12, fptype m13, fptype m23, unsigned int cyclic_index, fptype resMassSq, bool usenominal = true)
{
  if (0 == spin)
    return 1.0; // Should not cause branching since every thread evaluates the same resonance at the same time. 
  /*
  // Copied from BdkDMixDalitzAmp
   
  fptype _mA = (PAIR_12 == cyclic_index ? daug1Mass : (PAIR_13 == cyclic_index ? daug1Mass : daug3Mass)); 
  fptype _mB = (PAIR_12 == cyclic_index ? daug2Mass : (PAIR_13 == cyclic_index ? daug3Mass : daug3Mass)); 
  fptype _mC = (PAIR_12 == cyclic_index ? daug3Mass : (PAIR_13 == cyclic_index ? daug2Mass : daug1Mass)); 
    
  fptype _mAC = (PAIR_12 == cyclic_index ? m13 : (PAIR_13 == cyclic_index ? m12 : m12)); 
  fptype _mBC = (PAIR_12 == cyclic_index ? m23 : (PAIR_13 == cyclic_index ? m23 : m13)); 
  fptype _mAB = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23)); 

  // The above, collapsed into single tests where possible. 
  fptype _mA = (PAIR_13 == cyclic_index ? daug3Mass : daug2Mass);
  fptype _mB = (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass); 
  fptype _mC = (PAIR_12 == cyclic_index ? daug3Mass : (PAIR_13 == cyclic_index ? daug2Mass : daug1Mass)); 

  fptype _mAC = (PAIR_23 == cyclic_index ? m13 : m23);
  fptype _mBC = (PAIR_12 == cyclic_index ? m13 : m12);
  fptype _mAB = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23)); 
  */

  // Copied from EvtDalitzReso, with assumption that pairAng convention matches pipipi0 from EvtD0mixDalitz.
  // Again, all threads should get the same branch. 
  fptype _mA = (PAIR_12 == cyclic_index ? daug1Mass : (PAIR_13 == cyclic_index ? daug3Mass : daug2Mass));
  fptype _mB = (PAIR_12 == cyclic_index ? daug2Mass : (PAIR_13 == cyclic_index ? daug1Mass : daug3Mass));
  fptype _mC = (PAIR_12 == cyclic_index ? daug3Mass : (PAIR_13 == cyclic_index ? daug2Mass : daug1Mass));
  fptype _mAC = (PAIR_12 == cyclic_index ? m13 : (PAIR_13 == cyclic_index ? m23 : m12)); 
  fptype _mBC = (PAIR_12 == cyclic_index ? m23 : (PAIR_13 == cyclic_index ? m12 : m13)); 
  fptype _mAB = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
  return spinFactorABC(spin, motherMass, _mA, _mB, _mC, _mAB, _mAC, _mBC, usenominal ? 1.0/resMassSq : 1.0/_mAB);
}

__device__ devcomplex<fptype> plainBW (fptype m12, fptype m13, fptype m23, unsigned int* indices) {
  fptype motherMass             = functorConstants[indices[1]+0];
  fptype daug1Mass              = functorConstants[indices[1]+1];
  fptype daug2Mass              = functorConstants[indices[1]+2];
  fptype daug3Mass              = functorConstants[indices[1]+3];
  fptype meson_radius           = functorConstants[indices[1]+4];
  fptype mother_meson_radius    = functorConstants[indices[1]+5];
 
  fptype resmass                = hipArray[indices[2]];
  fptype reswidth               = hipArray[indices[3]];
  const unsigned int &spin                = indices[4];
  const unsigned int &cyclic_index        = indices[5];
  const unsigned int &use_nominal_resmass = indices[6];
  const unsigned int &use_unnorm_dampingf = indices[7];
  //printf("%.3f %.3f %.3f %.3f %.3f %.3f\n", motherMass, daug1Mass, daug2Mass, daug3Mass, meson_radius, mother_meson_radius);
  //printf("%.3f %.3f %d %d %d %d\n", resmass, reswidth, spin, cyclic_index, use_nominal_resmass, use_unnorm_dampingf);

  fptype rMassSq = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
  fptype frFactor(1.0);

  resmass *= resmass; 
  // Calculate momentum of the two daughters in the resonance rest frame; note symmetry under interchange (dm1 <-> dm2). 
  fptype measureDaughterMoms = twoBodyCMmom(rMassSq, 
					    (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass), 
					    (PAIR_12 == cyclic_index ? daug2Mass : daug3Mass));
  fptype nominalDaughterMoms = twoBodyCMmom(resmass, 
					    (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass), 
              (PAIR_12 == cyclic_index ? daug2Mass : daug3Mass));
  
  

  if (0 != spin)
    frFactor = dampingFactorRatioSquare(nominalDaughterMoms, measureDaughterMoms, spin, meson_radius, use_unnorm_dampingf);
 
  // RBW evaluation
  fptype A = (resmass - rMassSq); 
  fptype prat(measureDaughterMoms / nominalDaughterMoms);
  fptype B = resmass*reswidth * frFactor / SQRT(rMassSq);
  for(unsigned int i = 0; i < ((2*spin) + 1); ++i)
    B *= prat;
  fptype C = 1.0 / (A*A + B*B); 
  devcomplex<fptype> ret(A*C, B*C);
  // A + iB / (A^2 + B^2)
  
  // Don't want the F_D penetration factor in the mass-dependent width
  if(0 != spin)
  {
    fptype bachelorMass(PAIR_12 == cyclic_index ? daug3Mass : (PAIR_13 == cyclic_index ? daug2Mass : daug1Mass));
    frFactor *= dampingFactorRatioSquare(
        SQRT(FABS(bachelorMomSq(SQRT(resmass), motherMass, bachelorMass))),
        SQRT(FABS(bachelorMomSq(SQRT(rMassSq), motherMass, bachelorMass))),
        spin, mother_meson_radius, use_unnorm_dampingf); // using nominal/measured
  }

  ret *= SQRT(frFactor);
  fptype spinF = spinFactor(spin, motherMass, daug1Mass, daug2Mass, daug3Mass, m12, m13, m23, cyclic_index, resmass, use_nominal_resmass); 
  ret *= spinF; 

  //if(SQRT(resmass) < 0.900)
  //{
  //  printf("%.3f %.3f %.3f\n", frFactor, spinF, prat);
  // }

  return ret; 
}

// LASS shape for including a nonresonant effective-range component
// this is parameterised like BaBar arXiv:1004.5053
// http://arxiv.org/pdf/1004.5053v3.pdf
//
// but with some modifications to correct for the variations in normalisation.
// Their BW propagator has -- as most do -- dimensions of inverse mass squared, but their
// LASS propagator is dimensionless so we need to add the appropriate dimensioned
// constants to make these match when we sum them together
//
// The shape is supposed to be:
//
// sin(delta_R)*exp(i(delta_R + phi_R + 2delta_F + 2phi_F)) + F*sin(delta_F + phi_F)*exp(i(delta_F + phi_F))
//
// which we divide by exp(i*phi_R) and replace sin(delta_R)*exp(i*delta_R) by [BW], as evaluated by that implementation
// this latter part is what causes the dimensional problems, but I think it's the simplest way to implement this
//
// This leaves us with:
//
// [BW]*exp(2*i*(delta_F + phi_F)) + F*sin(delta_F + phi_F)*exp(i*(delta_F + phi_F - phi_R))
//
// with some dimensional constants multiplying the second term
// the use of the letter F is inheritied, B for background and N for nonresonant might both make sense...
//
// Update 20/06/2014 to try and match MINT more closely
//
// MINT seems to return (Lass2.C)
//
// (m_Kpi/q) * F*sin(phi_F + delta_F)*exp(i*(phi_F + delta_F)) + R*sin(delta_R) * exp(i*delta_R + 2*i*(delta_F + phi_F) + i*phi_R)
//
//
__device__ devcomplex<fptype> lass(fptype m12, fptype m13, fptype m23, unsigned int* indices) {
  // use the BW implementation for the resonant part...
  devcomplex<fptype> BW_part(plainBW(m12, m13, m23, indices));
  
  // need these to calculate the K,pi momentum in the Kpi frame
  fptype daug1Mass              = functorConstants[indices[1]+1];
  fptype daug2Mass              = functorConstants[indices[1]+2];
  fptype daug3Mass              = functorConstants[indices[1]+3];

  fptype resmass                = hipArray[indices[2]];
  fptype reswidth               = hipArray[indices[3]];
  //const unsigned int &spin                = indices[4];
  const unsigned int &cyclic_index        = indices[5];
  //const unsigned int &use_nominal_resmass = indices[6];
  //const unsigned int &use_unnorm_dampingf = indices[7];

  // extra LASS parameters
  fptype lass_a                 = hipArray[indices[8]];
  fptype lass_r                 = hipArray[indices[9]];
  fptype lass_phi_f             = hipArray[indices[10]];
  fptype lass_phi_r             = hipArray[indices[11]];
  fptype lass_F                 = hipArray[indices[12]];

  //printf("%.3f %.3f %.3f %.3f %.3f %d %.3f %.3f %.3f %.3f %.3f\n",
  //    daug1Mass, daug2Mass, daug3Mass, resmass, reswidth,
  //    cyclic_index, lass_a, lass_r, lass_phi_f, lass_phi_r, lass_F);
 
  fptype rMassSq((PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23)));
  fptype q(twoBodyCMmom(rMassSq,
                        (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass),
                        (PAIR_12 == cyclic_index ? daug2Mass : daug3Mass)));
  fptype q0(twoBodyCMmom(resmass*resmass,
                         (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass),
                         (PAIR_12 == cyclic_index ? daug2Mass : daug3Mass)));

  fptype delta_f(atan2(2.0*lass_a*q, 2.0 + (lass_a*lass_r*q*q)));
  fptype angle_combo(delta_f + lass_phi_f);
  
  BW_part *= devcomplex<fptype>(cos((2.0*angle_combo) + lass_phi_r), sin(2.0*(angle_combo) + lass_phi_r));
  BW_part *= reswidth * resmass * resmass / q0;
  
  devcomplex<fptype> nonres_part(lass_F * sin(angle_combo), 0);
  nonres_part *= devcomplex<fptype>(cos(angle_combo), sin(angle_combo));
  //fptype rMass(SQRT((PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23))));
  nonres_part *= SQRT(rMassSq) / q;
  // now correct for the way the other shapes are normalised
  // for whatever reason, Dalitz fitters seem to like dropping constants so that BW shapes have dimensions of inverse mass^2
  // but what we've just calculated is dimensionless
  // the magic quantity is, I think, this...
  //nonres_part *= q0 / (reswidth * resmass * resmass);
  
  return BW_part + nonres_part;
}

__device__ devcomplex<fptype> polylass(fptype m12, fptype m13, fptype m23, unsigned int *indices)
{
  // need these to calculate the K,pi momentum in the Kpi frame
  fptype motherMass             = functorConstants[indices[1]+0];
  fptype daug1Mass              = functorConstants[indices[1]+1];
  fptype daug2Mass              = functorConstants[indices[1]+2];
  fptype daug3Mass              = functorConstants[indices[1]+3];
  
  fptype resmass                = hipArray[indices[2]];
  fptype reswidth               = hipArray[indices[3]];
  unsigned int cyclic_index     = indices[5];
  // extra LASS parameters
  fptype lass_a                 = hipArray[indices[6]];
  fptype lass_r                 = hipArray[indices[7]];
  unsigned int num_poly_coeffs  = indices[8];
  unsigned int formfactor_type  = indices[9];
  // these are stored in hipArray[indices[9]] .. hipArray[indices[8 + num_poly_coeffs]]
  
  fptype rMassSq = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
  fptype q(twoBodyCMmom(rMassSq,
                        (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass),
                        (PAIR_12 == cyclic_index ? daug2Mass : daug3Mass)));
  fptype q0(twoBodyCMmom(resmass*resmass,
                         (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass),
                         (PAIR_12 == cyclic_index ? daug2Mass : daug3Mass)));
  
  // This is the LASS phase shift
  fptype delta_f(atan2(2.0*lass_a*q, 2.0 + lass_a*lass_r*q*q));
  
  // Calculate the running width
  fptype resrunwidth(reswidth * q * resmass / (q0 * SQRT(rMassSq)));
  // This is the regular BW phase shift
  fptype delta_r(atan2(resrunwidth * resmass, resmass * resmass - rMassSq));

  // To match the plainBW function we would have to return:
  //   (q0 * SQRT(rMassSq) / (reswidth * resmass * resmass * q)) * sin(delta_r) * exp(i*delta_r)
  // we want to do this but with delta_r -> delta_r + delta_f, and then multiply the whole thing by a polynomial
  
  devcomplex<fptype> ret(cos(delta_r + delta_f), sin(delta_r + delta_f));
  ret *= sin(delta_r + delta_f) * /*q0*/ SQRT(rMassSq) / q;//(reswidth * resmass * resmass * q);
  
  fptype poly(1.0);
  
  if(formfactor_type == ResonancePdf::RECURSIVEPOLY)
  {
    fptype coefffornext(1.0);
    fptype expansion_parameter(SQRT(rMassSq) / resmass);
    if(num_poly_coeffs == 0)
    {
      // If we don't have any floating parameters just return 1
    }
    else
    {
      poly = 0.0;

      // This should be a0*x*x + (1-a0)(a1*x + (1-a1))
      for(unsigned int poly_index = 0; poly_index <= num_poly_coeffs; ++poly_index)
      {
        fptype coeff(poly_index == num_poly_coeffs ? 1.0 : hipArray[indices[10 + poly_index]]);
        poly += pow(expansion_parameter, int(num_poly_coeffs - poly_index)) * coeff * coefffornext;
        coefffornext *= (1.0 - fabs(coeff));
     }
    }
  }
  else if((formfactor_type == ResonancePdf::NORMPOLY) || (formfactor_type == ResonancePdf::NORMEXPPOLY)
      || (formfactor_type == ResonancePdf::CENTRENORMEXPPOLY) || (formfactor_type == ResonancePdf::CENTRENORMEXPPOLYRECURSIVE)
      || (formfactor_type == ResonancePdf::CENTRENORMEXPPOLYDEEPRECURSIVE) || (formfactor_type == ResonancePdf::CENTRENORMEXPPOLYDEEPRECURSIVEALT))
  {
    fptype expansion_parameter(SQRT(rMassSq) / resmass);
    fptype _mA = (PAIR_12 == cyclic_index ? daug1Mass : (PAIR_13 == cyclic_index ? daug3Mass : daug2Mass));
    fptype _mB = (PAIR_12 == cyclic_index ? daug2Mass : (PAIR_13 == cyclic_index ? daug1Mass : daug3Mass));
    fptype _mC = (PAIR_12 == cyclic_index ? daug3Mass : (PAIR_13 == cyclic_index ? daug2Mass : daug1Mass));
    fptype centre_parameter(0.5*(motherMass - _mC + _mA + _mB) / resmass);
    fptype norm(1.0), centrenorm(1.0);
    poly = 1.0;
    for(unsigned int poly_index = 0; poly_index < num_poly_coeffs; ++poly_index)
    {
      fptype coeff(hipArray[indices[10 + poly_index]]);
      if(poly_index > 0 && formfactor_type == ResonancePdf::CENTRENORMEXPPOLYRECURSIVE)
        coeff *= hipArray[indices[10 + 0]]; // a,b,c -> (a, ab, ac)
      else if((formfactor_type == ResonancePdf::CENTRENORMEXPPOLYDEEPRECURSIVEALT)
          or (formfactor_type == ResonancePdf::CENTRENORMEXPPOLYDEEPRECURSIVE))
      {
        if(num_poly_coeffs != 3)
          printf("Wrong number of parameters for CENTRENORMEXPPOLYDEEPRECURSIVE[ALT]\n");
        fptype
          &c1(hipArray[indices[10 + 0]]),
          &c2(hipArray[indices[10 + 1]]),
          &c3(hipArray[indices[10 + 2]]);
        bool alt(formfactor_type == ResonancePdf::CENTRENORMEXPPOLYDEEPRECURSIVEALT);
        if(poly_index == 0)
        {
          if(alt)
            // neut
            coeff = -0.452*c1 -0.676*c2 - 0.582*c3;
          else
            coeff = -0.460*c1 + 0.702*c2 - 0.543*c3;
        }
        else if(poly_index == 1)
        {
          if(alt)
            // neut
            coeff = 0.776*c1 + 0.0243*c2 - 0.631*c3;
          else
            coeff = 0.776*c1 + 0.197*c2 - 0.631*c3;
        }
        else
        {
          if(alt)
            // neut
            coeff = -0.440*c1 + 0.737*c2 - 0.513*c3;
          else
            coeff = -0.433*c1 - 0.711*c2 - 0.554*c3;
        }
      }
      poly += pow(expansion_parameter, int(poly_index+1)) * coeff;
      norm += coeff;
      centrenorm += pow(centre_parameter, int(poly_index+1)) * coeff;
    }

    if(formfactor_type == ResonancePdf::NORMEXPPOLY)
    {
      poly = EXP(poly - norm); // exponential of the polynomial, divided by the exponential of when the parameter is 1
    }
    else if(formfactor_type == ResonancePdf::CENTRENORMEXPPOLY
        or formfactor_type == ResonancePdf::CENTRENORMEXPPOLYRECURSIVE
        or formfactor_type == ResonancePdf::CENTRENORMEXPPOLYDEEPRECURSIVE
        or formfactor_type == ResonancePdf::CENTRENORMEXPPOLYDEEPRECURSIVEALT)
    {
      poly = EXP(poly - centrenorm);
    }
    else
    {
      poly /= norm;
    }
  }
  else if(formfactor_type == ResonancePdf::POLY)
  {
    fptype expansion_parameter(SQRT(rMassSq) / resmass);
    poly = pow(expansion_parameter, int(num_poly_coeffs)); // 2 coefficents: a + bx + xx
    for(unsigned int poly_index = 0; poly_index < num_poly_coeffs; ++poly_index)
      poly += pow(expansion_parameter, int(poly_index)) * hipArray[indices[10 + poly_index]];
  }
  else if(formfactor_type == ResonancePdf::SENSIBLEPOLY)
  {
    // this expected to have as many coefficients as terms, and that the user will remember to fix one of them
    poly = 0.0;
    fptype expansion_parameter(SQRT(rMassSq) / resmass);
    for(unsigned int poly_index = 0; poly_index < num_poly_coeffs; ++poly_index)
      poly += hipArray[indices[10 + poly_index]] * pow(expansion_parameter, int(poly_index));
  }
  else if(formfactor_type == ResonancePdf::EXPPOLY)
  {
    // form factor f(x) = exp(g(x)) so f(x) > 0
    // g(x) expressed in Chebyshev polynomials
    // with x = SQRT(rMassSq) scaled to [-1,+1]
    fptype
      rMassMin(PAIR_12 == cyclic_index ? daug1Mass + daug2Mass : (PAIR_13 == cyclic_index ? daug1Mass + daug3Mass : daug2Mass + daug3Mass)),
      rMassMax(PAIR_12 == cyclic_index ? motherMass - daug3Mass: (PAIR_13 == cyclic_index ? motherMass - daug2Mass: motherMass - daug1Mass)),
      x(-1.0 + 2.0*(SQRT(rMassSq) - rMassMin)/(rMassMax - rMassMin)),
      norm(2.0);
    for(unsigned int poly_index = 1; poly_index <= num_poly_coeffs; ++poly_index)
    {
      fptype coeff(hipArray[indices[9 + poly_index]]);
      if(poly_index == 1)
        poly += coeff * x;
      else if(poly_index == 2)
      {
        poly += coeff * (2.0*x*x - 1.0);
        norm -= coeff * 2.0 / 3.0;
      }
      else if(poly_index == 3)
      {
        poly += coeff * (4.0*x*x*x - 3.0*x);
      }
      else
        printf("Too high an order requested from PolynomialLASS\n");  
    }
  }
  else
  {
    printf("Unknown form factor type requested\n");
  }
      
  // poly += pow(expansion_parameter, int(poly_index)) * hipArray[indices[8 + poly_index]];
  
  ret *= poly;
  return ret;
}

// this is the general case
__device__ devcomplex<fptype> flatte_rhohelper(fptype ma, fptype mb, fptype rMassSq)
{
  fptype tmp(1.0 - (ma - mb)*(ma - mb)/rMassSq);
  tmp *= (1.0 - (ma + mb)*(ma + mb)/rMassSq);
  if(tmp > 0.0)
    return devcomplex<fptype>(SQRT(tmp), 0);
  return devcomplex<fptype>(0, SQRT(-tmp));
}

// this is the specialised case when both masses are the same
__device__ devcomplex<fptype> flatte_rhohelper(fptype m, fptype rMassSq)
{
  fptype tmp(1.0 - (4.0*m*m/rMassSq));
  if(tmp > 0.0)
    return devcomplex<fptype>(SQRT(tmp), 0);
  return devcomplex<fptype>(0, SQRT(-tmp));
}

// coupled-channel lineshape for the a(0)(980) and f(0)(980) resonances
// quite which form it takes depends on which of the above we're dealing with, and also the charge
// a(0)(980) neutral: eta pi0, K+ K-, K0 K0
// a(0)(980) charged: eta pi+, K0 K+,
// f(0)(980) neutral: pi+ pi-, pi0 pi0, K+ K-, K0 K0
// f(0)(980) charged: pi+ pi0, K0 K+
//
// a(0)(980) has parameters g_EtaPi and g_KK
// f(0)(980) has parameters g_PiPi and g_KK (sometimes called g_Pi and g_K)
//
// In some limit of strong coupling, only the ratio of these is accessible, so deal with
// a(0)(980) g_EtaPi and g_KK/g_EtaPi
// f(0)(980) g_PiPi and g_KK/g_PiPi
// rather thean the two parameters separately
//
// Note the two g_KK are not the same for the two mesons, but these parameters are assumed to be
// the same for the neutral and charged versions
//
// These also seems to be disagreement over the convention for these couplings
// Either they can occur squared, or they can be multiplied by the nominal resonance mass
// to also produce a mass-dimension-2 quantity
//
// It's not clear which is better!
//
// We don't bother with any of the spin stuff because these two mesons are both scalars.
//
// Input parameters we need:
//   meson mass
//   cyclic_index
//   g_1
//   g_KK/g_1 -- interpreted as this squared depending on the other flags
//   whether we're squaring g_1 and g_KK or multiplying by resmass
//   whether we're talking about the a(0)(980) or the f(0)(980)
//   whether it's neutral or charged
//
// In the literature I've looked at, the first two flags aren't both neccessary, because we square the
// g_i in the a(0)(980) case but not the f(0)(980) case, but I don't understand why this must be the case
__device__ devcomplex<fptype> flatte(fptype m12, fptype m13, fptype m23, unsigned int* indices)
{
  fptype resmass                = hipArray[indices[2]];
  unsigned int cyclic_index     = indices[3];
  unsigned int square_couplings = indices[4];
  unsigned int a_meson          = indices[5];
  unsigned int charged_meson    = indices[6];
  fptype g_1                    = hipArray[indices[7]];
  fptype g_KK_over_g_1          = hipArray[indices[8]];
  
  fptype rMassSq = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
  const fptype mK0(0.497614), mK(0.493677), mPi(0.13957018), mPi0(0.1349766), mEta(0.547862);
  
  devcomplex<fptype> rho_1, rho_KK;
  
  if(a_meson)
  {
    if(charged_meson)
    {
      // charged a(0)(980)
      rho_1 = flatte_rhohelper(mEta, mPi, rMassSq); // eta pi+
      rho_KK = flatte_rhohelper(mK, mK0, rMassSq); // K0 K+
    }
    else
    {
      // neutral a(0)(980)
      rho_1 = flatte_rhohelper(mEta, mPi0, rMassSq); // eta pi0
      rho_KK = 0.5*(flatte_rhohelper(mK, rMassSq) + flatte_rhohelper(mK0, rMassSq)); // K+ K- and K0 K0
    }
  }
  else
  {
    if(charged_meson)
    {
      // charged f(0)(980)
      rho_1 = flatte_rhohelper(mPi, mPi0, rMassSq); // pi0 pi+
      rho_KK = flatte_rhohelper(mK, mK0, rMassSq); // K0 K+
    }
    else
    {
      // neutral f(0)(980)
      rho_1 = (1.0/3.0)*(2.0 * flatte_rhohelper(mPi, rMassSq) + flatte_rhohelper(mPi0, rMassSq)); // factor of 2 from isospin conservation
      rho_KK = 0.5*(flatte_rhohelper(mK, rMassSq) + flatte_rhohelper(mK0, rMassSq)); // K+ K- and K0 K0;
    }
  }
  
  // make g_i mass dimension 2 in the appropriate way
  if(square_couplings)
  {
    g_1 *= g_1;
    // in this case we interpret g_KK_over_g_1 as already being squared
    //g_KK_over_g_1 *= g_KK_over_g_1;
  }
  else
  {
    g_1 *= resmass;
    g_KK_over_g_1 *= resmass;
  }
  
  devcomplex<fptype> wid(rho_KK.real, rho_KK.imag); // not exactly a width, this has dimensions of mass^2
  wid *= g_KK_over_g_1;
  wid += rho_1;
  wid *= g_1;
  devcomplex<fptype> ret(resmass*resmass - rMassSq + wid.imag, wid.real);
  ret /= ret.abs2();
  
  return ret;
}

__device__ devcomplex<fptype> gaussian (fptype m12, fptype m13, fptype m23, unsigned int* indices) {
  // indices[1] is unused constant index, for consistency with other function types. 
  fptype resmass                = hipArray[indices[2]];
  fptype reswidth               = hipArray[indices[3]];
  unsigned int cyclic_index     = indices[4]; 

  // Notice sqrt - this function uses mass, not mass-squared like the other resonance types. 
  fptype massToUse = SQRT(PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
  massToUse -= resmass;
  massToUse /= reswidth;
  massToUse *= massToUse;
  fptype ret = EXP(-0.5*massToUse); 

  // Ignore factor 1/sqrt(2pi). 
  ret /= reswidth;

  return devcomplex<fptype>(ret, 0); 
}

// Gou-Sak h()
__device__ fptype hFun (double s, double daug2Mass, double daug3Mass) {
  // Last helper function
  const fptype _pi = 3.14159265359;
  double sm   = daug2Mass + daug3Mass;
  double SQRTs = sqrt(s);
  double k_s = twoBodyCMmom(s, daug2Mass, daug3Mass);

  double val = ((2/_pi) * (k_s/SQRTs) * log( (SQRTs + 2*k_s)/(sm)));

  return val;
}

// Gou-Sak h'()
// This is strictly correct (the derivative of h(s) w.r.t 's') only when the daughter masses are equal
__device__ fptype dh_dsFun (double s, double daug2Mass, double daug3Mass) {
  // Yet another helper function
  const fptype _pi = 3.14159265359;
  double k_s = twoBodyCMmom(s, daug2Mass, daug3Mass);
  
  double val = (hFun(s, daug2Mass, daug3Mass) * (1.0/(8.0*pow(k_s, 2)) - 1.0/(2.0 * s)) + 1.0/(2.0* _pi*s));
  return val;
}


__device__ fptype dFun (double s, double daug2Mass, double daug3Mass) {
  // Helper function used in Gronau-Sakurai
  const fptype _pi = 3.14159265359;
  double sm   = daug2Mass + daug3Mass;
  double sm24 = sm*sm/4.0; // average daughter mass squared
  double m    = sqrt(s); // nominal resonance mass
  double k_m2 = twoBodyCMmom(s, daug2Mass, daug3Mass); // this is the nominal momentum
  double val = 3.0/_pi * sm24/pow(k_m2, 2) * log((m + 2*k_m2)/sm) + m/(2*_pi*k_m2) - sm24*m/(_pi * pow(k_m2, 3));
  return val;
}

__device__ fptype fsFun (double s, double m2, double gam, double daug2Mass, double daug3Mass) {
  // Another G-S helper function
   
  double k_s   = twoBodyCMmom(s,  daug2Mass, daug3Mass);
  double k_Am2 = twoBodyCMmom(m2, daug2Mass, daug3Mass);
   
  double f     = gam * m2 / POW(k_Am2, 3);
  f           *= (POW(k_s, 2) * (hFun(s, daug2Mass, daug3Mass) - hFun(m2, daug2Mass, daug3Mass)) + (m2 - s) * pow(k_Am2, 2) * dh_dsFun(m2, daug2Mass, daug3Mass));
  // dh_dsFun is the h' function of the original paper

  return f;
}

__device__ devcomplex<fptype> gouSak (fptype m12, fptype m13, fptype m23, unsigned int* indices) {
  fptype motherMass             = functorConstants[indices[1]+0];
  fptype daug1Mass              = functorConstants[indices[1]+1];
  fptype daug2Mass              = functorConstants[indices[1]+2];
  fptype daug3Mass              = functorConstants[indices[1]+3];
  fptype meson_radius           = functorConstants[indices[1]+4];
  fptype mother_meson_radius    = functorConstants[indices[1]+5];

  fptype resmass                = hipArray[indices[2]];
  fptype resmassSq              = resmass*resmass;
  fptype reswidth               = hipArray[indices[3]];
  const unsigned int &spin                = indices[4];
  const unsigned int &cyclic_index        = indices[5]; 
  const unsigned int &use_nominal_mass    = indices[6];
  const unsigned int &use_unnorm_dampingf = indices[7];

  fptype rMassSq = (PAIR_12 == cyclic_index ? m12 : (PAIR_13 == cyclic_index ? m13 : m23));
  fptype daugAMass = (PAIR_23 == cyclic_index ? daug2Mass : daug1Mass); // These are the two daughters
  fptype daugBMass = (PAIR_12 == cyclic_index ? daug2Mass : daug3Mass); // of the resonance
  fptype frFactor = 1;

  // Calculate momentum of the two daughters in the resonance rest frame; note symmetry under interchange (dm1 <-> dm2). 
  fptype measureDaughterMoms = twoBodyCMmom(rMassSq, daugAMass, daugBMass);
  fptype nominalDaughterMoms = twoBodyCMmom(resmassSq, daugAMass, daugBMass);

  if (0 != spin)
  {
    frFactor = dampingFactorRatioSquare(
        nominalDaughterMoms,
        measureDaughterMoms,
        spin, meson_radius,
        use_unnorm_dampingf);
  }

  fptype runningwidth = reswidth * frFactor * (resmass/SQRT(rMassSq)) * POW(measureDaughterMoms / nominalDaughterMoms, 2.0*spin+1.0);
  
  // Implement Gou-Sak:
  fptype D = (1.0 + dFun(resmassSq, daugAMass, daugBMass)*reswidth/resmass);
  fptype E = resmassSq - rMassSq + fsFun(rMassSq, resmassSq, reswidth, daugAMass, daugBMass);
  fptype F = resmass * runningwidth;
  //SQRT(resmass) * reswidth * POW(measureDaughterMoms / nominalDaughterMoms, 2.0*spin + 1) * frFactor;

  D /= (E*E + F*F);
  devcomplex<fptype> retur(D*E, D*F);

  // Didn't want the F_D penetration factor in the mass-dependent width, but we do want it in the overall spin factor
  if(0 != spin)
  {
    fptype bachelorMass(PAIR_12 == cyclic_index ? daug3Mass : (PAIR_13 == cyclic_index ? daug2Mass : daug1Mass));
    frFactor *= dampingFactorRatioSquare(
        bachelorMom(resmass, motherMass, bachelorMass),
        bachelorMom(SQRT(rMassSq), motherMass, bachelorMass),
        spin, mother_meson_radius,
        use_unnorm_dampingf);
  }

  retur *= SQRT(frFactor);
  retur *= spinFactor(spin, motherMass, daug1Mass, daug2Mass, daug3Mass, m12, m13, m23, cyclic_index, resmassSq, use_nominal_mass);
  return retur; 
}

__device__ devcomplex<fptype> nonres (fptype m12, fptype m13, fptype m23, unsigned int* indices) {
  return devcomplex<fptype>(1, 0); 
}


__device__ void getAmplitudeCoefficients (devcomplex<fptype> a1, devcomplex<fptype> a2, fptype& a1sq, fptype& a2sq, fptype& a1a2real, fptype& a1a2imag) {
  // Returns A_1^2, A_2^2, real and imaginary parts of A_1A_2^*
  a1sq = a1.abs2();
  a2sq = a2.abs2();
  a1 *= conj(a2);
  a1a2real = a1.real;
  a1a2imag = a1.imag; 
}

__device__ resonance_function_ptr ptr_to_RBW = plainBW;
__device__ resonance_function_ptr ptr_to_FLATTE = flatte;
__device__ resonance_function_ptr ptr_to_LASS = lass;
__device__ resonance_function_ptr ptr_to_polynomialLASS = polylass;
__device__ resonance_function_ptr ptr_to_GOUSAK = gouSak; 
__device__ resonance_function_ptr ptr_to_GAUSSIAN = gaussian;
__device__ resonance_function_ptr ptr_to_NONRES = nonres;


ResonancePdf::ResonancePdf (string name,
    const AmplitudeInfo &amp_,
    Variable* mass, 
    Variable* width, 
    unsigned int sp, 
    unsigned int cyc,
    bool useNominalMass,
    bool useUnNormalisedDampingFactors) 
  : GooPdf(0, name), amp(amp_)
{
  vector<unsigned int> pindices; 
  pindices.push_back(0); 
  // Making room for index of decay-related constants. Assumption:
  // These are mother mass and three daughter masses in that order.
  // They will be registered by the object that uses this resonance,
  // which will tell this object where to find them by calling setConstantIndex. 

  pindices.push_back(registerParameter(mass));
  pindices.push_back(registerParameter(width)); 
  pindices.push_back(sp);
  pindices.push_back(cyc); 
  pindices.push_back(useNominalMass);
  pindices.push_back(useUnNormalisedDampingFactors);

  hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_RBW, sizeof(void*));
  initialise(pindices); 
}

ResonancePdf::ResonancePdf (string name,
    const AmplitudeInfo &amp_,
    Variable* mass,
    Variable* g_1,
    Variable* g_KK_over_g_1,
    unsigned int cyc,
    CouplingTreatment square_couplings,
    WhichMeson a_meson,
    MesonCharge charged_meson)
: GooPdf(0, name), amp(amp_)
{
  vector<unsigned int> pindices;
  pindices.push_back(0);
  // Making room for index of decay-related constants. Assumption:
  // These are mother mass and three daughter masses in that order.
  // They will be registered by the object that uses this resonance,
  // which will tell this object where to find them by calling setConstantIndex.

  pindices.push_back(registerParameter(mass));
  pindices.push_back(cyc);
  pindices.push_back(square_couplings);
  pindices.push_back(a_meson);
  pindices.push_back(charged_meson);
  pindices.push_back(registerParameter(g_1));
  pindices.push_back(registerParameter(g_KK_over_g_1));
  
  hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_FLATTE, sizeof(void*));
  initialise(pindices); 
}

ResonancePdf::ResonancePdf (string name,
    const AmplitudeInfo &amp_,
    Variable* mass,
    Variable* width,
    Variable* lass_a,
    Variable* lass_r,
    Variable* lass_phi_f,
    Variable* lass_phi_r,
    Variable* lass_F,
    unsigned int sp,
    unsigned int cyc)
: GooPdf(0, name), amp(amp_)
{
  vector<unsigned int> pindices;
  pindices.push_back(0);
  // Making room for index of decay-related constants. Assumption:
  // These are mother mass and three daughter masses in that order.
  // They will be registered by the object that uses this resonance,
  // which will tell this object where to find them by calling setConstantIndex.
  
  pindices.push_back(registerParameter(mass));
  pindices.push_back(registerParameter(width));
  pindices.push_back(sp);
  pindices.push_back(cyc);
  pindices.push_back(0);// useNominalMass -- this lets us defer to plainBW()
  pindices.push_back(0);// useUnNormalisedDampingFactors -- this lets us defer to plainBW()
  pindices.push_back(registerParameter(lass_a));
  pindices.push_back(registerParameter(lass_r));
  pindices.push_back(registerParameter(lass_phi_f));
  pindices.push_back(registerParameter(lass_phi_r));
  pindices.push_back(registerParameter(lass_F));

  
  hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_LASS, sizeof(void*));
  initialise(pindices);
  
  if(sp)
    std::cout << "WARNING from LASS shape constructor: spin " << sp << " requested, at present this is ignored" << std::endl;
}

ResonancePdf::ResonancePdf(std::string name,
    const AmplitudeInfo &amp_,
    Variable* mass,
    Variable* width,
    Variable* lass_a,
    Variable* lass_r,
    const std::vector<Variable*> &poly_coeffs,
    unsigned int sp,
    unsigned int cyc,
    FormFactorType fftype)
: GooPdf(0, name), amp(amp_)
{
  std::vector<unsigned int> pindices;
  pindices.push_back(0); // copying the rest of the constructors...
  pindices.push_back(registerParameter(mass));
  pindices.push_back(registerParameter(width));
  pindices.push_back(sp);
  pindices.push_back(cyc);
  pindices.push_back(registerParameter(lass_a));
  pindices.push_back(registerParameter(lass_r)); // might as well match the other LASS shape up to this point
  pindices.push_back(poly_coeffs.size());
  pindices.push_back(fftype);
  for(std::vector<Variable*>::const_iterator poly_iter = poly_coeffs.begin(); poly_iter != poly_coeffs.end(); poly_iter++)
    pindices.push_back(registerParameter(*poly_iter));
  
  hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_polynomialLASS, sizeof(void*));
  initialise(pindices);
  
  if(sp)
    std::cout << "WARNING from Polynomial LASS shape constructor: spin " << sp << " requested, at present this is ignored" << std::endl;
}

ResonancePdf::ResonancePdf (string name,
    const AmplitudeInfo &amp_,
    unsigned int sp, 
    Variable* mass, 
    Variable* width, 
    unsigned int cyc,
    bool useNominalMass,
    bool useUnNormalisedDampingFactors) 
: GooPdf(0, name), amp(amp_)
{
  // Same as BW except for function pointed to. 
  vector<unsigned int> pindices; 
  pindices.push_back(0); 
  pindices.push_back(registerParameter(mass));
  pindices.push_back(registerParameter(width)); 
  pindices.push_back(sp);
  pindices.push_back(cyc);
  pindices.push_back(useNominalMass);
  pindices.push_back(useUnNormalisedDampingFactors);

  hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_GOUSAK, sizeof(void*));
  initialise(pindices); 
}

ResonancePdf::ResonancePdf (string name, const AmplitudeInfo &amp_) 
  : GooPdf(0, name), amp(amp_)
{
  vector<unsigned int> pindices; 
  pindices.push_back(0); 
  // Dummy index for constants - won't use it, but calling 
  // functions can't know that and will call setConstantIndex anyway. 
  hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_NONRES, sizeof(void*));
  initialise(pindices); 
}

ResonancePdf::ResonancePdf (string name,
    const AmplitudeInfo &amp_,
    Variable* mean, 
    Variable* sigma,
    unsigned int cyc) 
  : GooPdf(0, name), amp(amp_)
{
  vector<unsigned int> pindices; 
  pindices.push_back(0); 
  // Dummy index for constants - won't use it, but calling 
  // functions can't know that and will call setConstantIndex anyway. 
  pindices.push_back(registerParameter(mean));
  pindices.push_back(registerParameter(sigma)); 
  pindices.push_back(cyc); 

  hipMemcpyFromSymbol((void**) &host_fcn_ptr, ptr_to_GAUSSIAN, sizeof(void*));
  initialise(pindices); 	
}
